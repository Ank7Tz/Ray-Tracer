#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

template <typename T>
void check(T err, const char* const func, const char* file, const int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, 
            line, static_cast<unsigned int>(err), hipGetErrorString(err), 
            func);
        exit(EXIT_FAILURE);
    }
}

__global__ void generate_frame(unsigned int *buffer, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;

    if (idx < total_pixels) {
        int i = idx % width;
        int j = idx / width;

        float r = float(i) / (width - 1);
        float g = float(j) / (height - 1);
        float b = 0.0f;

        int buffer_idx = idx * 3;
        buffer[buffer_idx] = int(255.999 * r);
        buffer[buffer_idx + 1] = int(255.999 * g);
        buffer[buffer_idx + 2] = int(255.999 * b);
    }
}

int main() {
    int image_width = 4096;
    int image_height = 2160;
    int total_pixels = image_width * image_height;
    int buffer_size = total_pixels * 3;
    hipEvent_t start, stop;
    hipStream_t stream1;

    int total_blocks = (total_pixels + BLOCK_SIZE - 1) / BLOCK_SIZE; // (256 * 256 + 128 - 1) / 256 = 256 blocks

    unsigned int *frame_buffer; // The Host's frame buffer.

    unsigned int *device_frame_buffer; // Device's frame buffer.

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    CHECK_CUDA_ERROR(hipStreamCreate(&stream1));
    
    CHECK_CUDA_ERROR(hipEventRecord(start, stream1));
    
    CHECK_CUDA_ERROR(hipHostMalloc(&frame_buffer, sizeof(int) * buffer_size, hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipMallocAsync(&device_frame_buffer, sizeof(int) * buffer_size, stream1));

    generate_frame<<<total_blocks, BLOCK_SIZE, 0, stream1>>>(device_frame_buffer, image_width, image_height);

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));

    CHECK_CUDA_ERROR(hipMemcpyAsync(frame_buffer, device_frame_buffer, sizeof(int) * buffer_size, hipMemcpyDeviceToHost, stream1));

    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    for (int j = 0; j < image_height; j++) {
        for (int i = 0; i < image_width; i++) {
            int idx = (j * image_width + i) * 3;
            std::cout << frame_buffer[idx] << ' ' << frame_buffer[idx + 1] << ' ' << frame_buffer[idx + 2] << '\n';
        }
    }

    CHECK_CUDA_ERROR(hipEventRecord(stop, stream1));

    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    float milliseconds = 0;

    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));

    CHECK_CUDA_ERROR(hipHostFree(frame_buffer));
    CHECK_CUDA_ERROR(hipFree(device_frame_buffer));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream1));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    std::cerr << "Execution time: " << milliseconds << " milliseconds" << std::endl;

    return 0;
}