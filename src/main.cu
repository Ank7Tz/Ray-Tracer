#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "vec3.h"
#include "color.h"
#include "ray.h"

#define BLOCK_SIZE 256

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

template <typename T>
void check(T err, const char* const func, const char* file, const int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, 
            line, static_cast<unsigned int>(err), hipGetErrorString(err), 
            func);
        exit(EXIT_FAILURE);
    }
}

__constant__ vec3 g_pixel00_loc;
__constant__ vec3 g_camera_center;
__constant__ vec3 g_pixel_delta_u;
__constant__ vec3 g_pixel_delta_v;

__host__ __device__ color ray_color(const ray& r) {
    vec3 unit_direction = unit_vector(r.direction());
    auto a = 0.5 * (unit_direction.y() + 1.0);
    return (1.0 - a) * color(1.0, 1.0, 1.0) + a * color(0.5, 0.7, 1.0);
}

__global__ void generate_frame(unsigned int *buffer, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;

    if (idx < total_pixels) {
        int i = idx % width;
        int j = idx / width;
        int buffer_idx = idx * 3;

        auto pixel_center = g_pixel00_loc + (i * g_pixel_delta_u) + (j * g_pixel_delta_v);
        auto ray_direction = pixel_center - g_camera_center;
        ray r(g_camera_center, ray_direction);

        color pixel_color = ray_color(r);

        write_color(buffer, buffer_idx, pixel_color);
    }
}


int main() {
    // Image
    auto aspect_ratio = 4096.00 / 2160.00;
    int image_width = 4096;
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;
    int total_pixels = image_width * image_height;

    // Camera
    auto focal_length = 1.0;
    auto viewport_height = 2.0;
    auto viewport_width = viewport_height * 
                            (double(image_width) / image_height);
    auto camera_center = point3(0, 0, 0);

    // viewport vectors
    auto viewport_u = vec3(viewport_width, 0, 0);
    auto viewport_v = vec3(0, -viewport_height, 0);

    // pixel delta
    auto pixel_delta_u = viewport_u / image_width;
    auto pixel_delta_v = viewport_v / image_height;

    // viewport upper left corner
    auto viewport_upper_left = camera_center
                                - vec3(0, 0, -focal_length)
                                - (viewport_u / 2)
                                - (viewport_v / 2);

    auto pixel00_loc = viewport_upper_left
                        + (0.5 * (pixel_delta_u + pixel_delta_v));
    
    int buffer_size = total_pixels * 3;
    hipEvent_t start, stop;
    hipStream_t stream1;

    // (256 * 256 + 128 - 1) / 256 = 256 blocks
    int total_blocks = (total_pixels + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // The Host's frame buffer.
    unsigned int *frame_buffer;

    // Device's frame buffer.
    unsigned int *device_frame_buffer;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    CHECK_CUDA_ERROR(hipStreamCreate(&stream1));
    
    CHECK_CUDA_ERROR(hipEventRecord(start, stream1));

    // Copy copy constant values to device memory
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(g_pixel00_loc), &pixel00_loc, sizeof(pixel00_loc)));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(g_camera_center), &camera_center, sizeof(camera_center)));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(g_pixel_delta_u), &pixel_delta_u, sizeof(pixel_delta_u)));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(g_pixel_delta_v), &pixel_delta_v, sizeof(pixel_delta_v)));

    
    CHECK_CUDA_ERROR(hipHostMalloc(&frame_buffer, sizeof(int) * buffer_size));
    CHECK_CUDA_ERROR(hipMallocAsync(&device_frame_buffer, sizeof(int) * buffer_size, stream1));

    generate_frame<<<total_blocks, BLOCK_SIZE, 0, stream1>>>(device_frame_buffer, image_width, image_height);

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));

    CHECK_CUDA_ERROR(hipMemcpyAsync(frame_buffer, device_frame_buffer, sizeof(int) * buffer_size, hipMemcpyDeviceToHost, stream1));

    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    for (int j = 0; j < image_height; j++) {
        for (int i = 0; i < image_width; i++) {
            int idx = (j * image_width + i) * 3;
            std::cout << frame_buffer[idx] << ' ' << frame_buffer[idx + 1] << ' ' << frame_buffer[idx + 2] << '\n';
        }
    }

    CHECK_CUDA_ERROR(hipEventRecord(stop, stream1));

    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    float milliseconds = 0;

    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));

    CHECK_CUDA_ERROR(hipHostFree(frame_buffer));
    CHECK_CUDA_ERROR(hipFree(device_frame_buffer));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream1));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    std::clog << "Execution time: " << milliseconds << " milliseconds" << std::endl;

    return 0;
}